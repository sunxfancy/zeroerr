#include "hip/hip_runtime.h"
#define ZEROERR_ENABLE_PFR
#include "zeroerr/assert.h"
#include "zeroerr/dbg.h"
#include "zeroerr/print.h"
#include "zeroerr/unittest.h"
#include "zeroerr/hip/hip_runtime.h"

#include <assert.h>
#include <hip/hip_runtime.h>

#define gpuErrchk(ans) \
    { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

#define N 1024
__global__ void add(int *a, int *b, int *c, unsigned n) {
    unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n) {
        c[id] = a[id] + b[id];
    }
}

void random_ints(int *a, int k) {
    for (int i = 0; i < k; ++i) a[i] = rand();
}


TEST_CASE("CUDA basic test") {
    int *a, *b, *c;        // host copies of a, b, c
    int *d_a, *d_b, *d_c;  // device copies of a, b, c
    int  size = N * sizeof(int);
    // Allocate space for device copies of a, b, c
    gpuErrchk(hipMalloc((void **)&d_a, size));
    gpuErrchk(hipMalloc((void **)&d_b, size));
    gpuErrchk(hipMalloc((void **)&d_c, size));
    // Setup input values
    a = (int *)malloc(size);
    random_ints(a, N);
    b = (int *)malloc(size);
    random_ints(b, N);
    c = (int *)malloc(size);

    // Copy inputs to device
    gpuErrchk(hipMemcpy(d_a, a, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_b, b, size, hipMemcpyHostToDevice));
    // Launch add() kernel on GPU
    add<<<N / 32, 32>>>(d_a, d_b, d_c, N);
    gpuErrchk(hipGetLastError());
    gpuErrchk(hipDeviceSynchronize());
    // Copy result back to host
    gpuErrchk(hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost));

    dbg(c);
    for (int i = 0; i < N; ++i) {
        CHECK(c[i] == (a[i] + b[i]));
    }

    // Cleanup
    free(a);
    free(b);
    free(c);
    gpuErrchk(hipFree(d_a));
    gpuErrchk(hipFree(d_b));
    gpuErrchk(hipFree(d_c));
}